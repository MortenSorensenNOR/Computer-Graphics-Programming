#include "hip/hip_runtime.h"
#include <stdio.h>
#include "rasterizer.h"

__global__ void VecAdd(float* A, float* B, float* C, int num_elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < num_elements) {
        C[i] = A[i] + B[i];
    }
}
